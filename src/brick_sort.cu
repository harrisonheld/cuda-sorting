#include "hip/hip_runtime.h"
#include "sort_kernels.cuh"
#include <hip/hip_runtime.h>

#define THREADS 256

__global__ void odd_even_kernel(int* arr, size_t n, int phase, bool* d_swapped) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Odd phase: start = 1, Even phase: start = 0
    size_t idx = 2 * i + phase;
    if (idx + 1 >= n) return;

    int a = arr[idx];
    int b = arr[idx + 1];
    if (a > b) {
        arr[idx] = b;
        arr[idx + 1] = a;
        *d_swapped = true;
    }
}

void brick_sort(int* arr, size_t n) {
    if (n < 2) return;

    int* d_arr;
    bool* d_swapped;
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_swapped, sizeof(bool));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    size_t blocks = (n + THREADS - 1) / (2 * THREADS);

    bool h_swapped = true;
    while (h_swapped) {
        h_swapped = false;
        hipMemcpy(d_swapped, &h_swapped, sizeof(bool), hipMemcpyHostToDevice);

        // odd indices (1, 3, 5, ...)
        odd_even_kernel<<<blocks, THREADS>>>(d_arr, n, 1, d_swapped);
        hipDeviceSynchronize();

        // even indices (0, 2, 4, ...)
        odd_even_kernel<<<blocks, THREADS>>>(d_arr, n, 0, d_swapped);
        hipDeviceSynchronize();

        hipMemcpy(&h_swapped, d_swapped, sizeof(bool), hipMemcpyDeviceToHost);
    }

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
    hipFree(d_swapped);
}
